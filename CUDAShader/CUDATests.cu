#include "hip/hip_runtime.h"
#include "CUDATests.cuh"

#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <chrono> // A SUPPRIMER

#define USING_GPU true
#define N 10

__declspec(dllexport) void tests() {
    using std::cout;
    using std::endl;

    int nb_exec = 50;

    for (int i = 0; i < nb_exec; i++) {
        std::chrono::steady_clock::time_point start = std::chrono::high_resolution_clock::now();
        mainTestCUDA();
        std::chrono::steady_clock::time_point stop = std::chrono::high_resolution_clock::now();
        std::chrono::microseconds duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        char* device_name = USING_GPU ? "GPU" : "CPU";
        cout << "Temps de calcul (" << device_name << "): " << duration.count() / 1000. << "ms" << endl;
    }
}

void mainTestCUDA() {
    using std::cout;
    using std::endl;

    // Allocations et initialisations
    int* a, * b, * c;
    hipMallocManaged(&a, N * sizeof(int));
    hipMallocManaged(&b, N * sizeof(int));
    hipMallocManaged(&c, N * sizeof(int));
    assert(a && b && c);

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = N * i; // N fois plus
        c[i] = 0;
    }

    // Calcul (soit GPU, soit CPU)
    if (USING_GPU) vectorAddCUDA<<<1, N >>>(a, b, c);
    else vectorAdd(a, b, c);
    hipDeviceSynchronize();

    // Lib�ration de m�moire
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

void vectorAdd(int* a, int* b, int* c) {
    int i;
    for (i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

__global__ void vectorAddCUDA(int* a, int* b, int* c) {
    int i = threadIdx.x;
    if (i < N) c[i] = a[i] + b[i];
}