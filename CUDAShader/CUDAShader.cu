#include "hip/hip_runtime.h"
﻿#include "CUDAShader.cuh"

#include <iostream>
#include <stdio.h>
#include <assert.h>

/**
* Notes:
* Dans les propriétés du projet CUDAShader -> Cuda C/C++ -> Device -> Code Generation
* La valeur par défaut est "compute_52,sm_52"
* Je l'ai changée pour "compute_75,sm_75"
* Car ma RTX 2080Ti est d'architecture Turing, et supporte donc jusqu'au sm_75
* Les constantes ci-dessous sont définies en fonction de cette architecture
*/

// Grid ; max grid dimension = 3
#define THEORATICAL_MAX_X_GRID_DIMENSION 2147483647 // 2^31-1 ; works fine
#define THEORATICAL_MAX_YZ_GRID_DIMENSION 65535     // 2^16 ; works fine
// Block ; max block dimension = 3
#define THEORETICAL_MAX_XY_BLOCK_DIMENSION 1024     // 2^10 ; real limit is lower ?
#define THORETICAL_MAX_Z_BLOCK_DIMENSION 64
// Threads
#define THEORETICAL_MAX_THREADS_PER_BLOCK 1024

#define MAX_X_BLOCK_DIMENSION 896 // Limite trouvée par tests, pour tout le reste à 1

__declspec(dllexport) int** allocate_tab(int n, int m) {
    int** res;
    res = (int**)malloc(n * sizeof(int*));

    if (res == NULL) {
        perror("allocate_tab");
        return NULL;
    }

    for (int i = 0; i < n; i++) {
        res[i] = (int*)malloc(m * sizeof(int));

        if (res[i] == NULL) {
            perror("allocate_tab");
            return NULL;
        }
    }
    return res;
}

__declspec(dllexport) void free_tab(int** tab, int n, int m) {
    if (tab == NULL) return;

    for (int i = 0; i < n; i++) {
        if (tab[i] == NULL) continue;
        free(tab[i]);
    }

}

__declspec(dllexport) void calculate_Julia(double c_re, double c_im, int** h_res, int width, int height, double scale, double offset_re, double offset_im) {
    hipError_t cuda_res;
    int nbPixel = width * height;
    int* device_tab;
    
    // Allocate device memory (no need to init)
    // Using one dimensionnal array (seems to be better for cuda programming)
    cuda_res = hipMallocManaged(&device_tab, nbPixel * sizeof(int));
    if (cuda_res != hipSuccess) { perror("hipMallocManaged"); exit(EXIT_FAILURE); }

    // Determining dimension to execute
    int block_dim = MAX_X_BLOCK_DIMENSION; // we're using only on dimensionnal blocks (since we cannot go ver 896, for no found reason...)
    int grid_dim = nbPixel / MAX_X_BLOCK_DIMENSION;
    dim3 grid_size(grid_dim); // aka number of blocks (per grid)
    dim3 block_size(block_dim); // aka number of thread (per blocks)

    // Executing kernel
    calculate_Julia_GPU<<<grid_size, block_size>>>(c_re, c_im, device_tab, nbPixel, width, height, scale, offset_re, offset_im);
    cuda_res = hipDeviceSynchronize();
    if (cuda_res != hipSuccess) { 
        std::cout << "\n" << cuda_res << std::endl;
        perror("hipDeviceSynchronize"); exit(EXIT_FAILURE); 
    }

    // Copy results from device to host
    for (int i = 0; i < nbPixel; i++) h_res[i / height][i % height] = device_tab[i];

    // Free allocated memory
    hipFree(device_tab);
}

__declspec(dllexport) void new_calculate_Julia(double c_re, double c_im, int** h_res, int width, int height, double x_min, double y_min, double x_max, double y_max) {
    hipError_t cuda_res;
    int nbPixel = width * height;
    int* device_tab;

    // Allocate device memory (no need to init)
    // Using one dimensionnal array (seems to be better for cuda programming)
    cuda_res = hipMallocManaged(&device_tab, nbPixel * sizeof(int));
    if (cuda_res != hipSuccess) { perror("hipMallocManaged"); exit(EXIT_FAILURE); }

    // Determining dimension to execute
    int block_dim = MAX_X_BLOCK_DIMENSION; // we're using only on dimensionnal blocks (since we cannot go ver 896, for no found reason...)
    int grid_dim = nbPixel / MAX_X_BLOCK_DIMENSION;
    dim3 grid_size(grid_dim); // aka number of blocks (per grid)
    dim3 block_size(block_dim); // aka number of thread (per blocks)

    // Executing kernel
    new_calculate_Julia_GPU<<<grid_size, block_size>>>(c_re, c_im, device_tab, nbPixel, width, height, x_min, y_min, x_max, y_max);
    cuda_res = hipDeviceSynchronize();
    if (cuda_res != hipSuccess) {
        std::cout << "\n" << cuda_res << std::endl;
        perror("hipDeviceSynchronize"); exit(EXIT_FAILURE);
    }

    // Copy results from device to host
    for (int i = 0; i < nbPixel; i++) h_res[i / height][i % height] = device_tab[i];

    // Free allocated memory
    hipFree(device_tab);
}

__global__ void calculate_Julia_GPU(double c_re, double c_im, int* d_res, const int nbPixel, const int width, const int height, double scale, double offset_re, double offset_im) {
    int n = blockDim.x * blockIdx.x + threadIdx.x; // index of thread: block_idx * 896 + thread_idx     // For fullHD: blockIdx € [0,2315] and threadIdx € [0,896] so n < 2315*896
    int i = n / height;
    int j = n % height;

    if (!(n < nbPixel)) return;

    double z0_re = (i - offset_re) * scale;
    double z0_im = (j - offset_im) * scale;

    // isSuiteDivergente

    double zi_re = z0_re; // init zi = z0
    double zi_im = z0_im;
    double zi_mod = z0_re * z0_re + z0_im * z0_im;

    double zi_re_temp;
    const unsigned int MAX_ITERATION = 200;
    const double MODULE_MAX = 4.0;

    int nb_iteration = 0;
    while (zi_mod < MODULE_MAX && nb_iteration < MAX_ITERATION) {
        zi_re_temp = zi_re;                             // next zi
        zi_re = zi_re * zi_re - zi_im * zi_im + c_re;   // next zi
        zi_im = 2 * zi_re_temp * zi_im + c_im;          // next zi
        zi_mod = zi_re * zi_re + zi_im * zi_im;   // update zi (squared) module
        nb_iteration++;
    }

    if (zi_mod >= MODULE_MAX) { // On a un module très grand
        d_res[i * height + j] = true;
    }
    else {
        d_res[i * height + j] = false;
    }

    // fin isSuiteDivergente

}

__global__ void new_calculate_Julia_GPU(double c_re, double c_im, int* d_res, const int nbPixel, const int width, const int height, double x_min, double y_min, double x_max, double y_max) {
    int n = blockDim.x * blockIdx.x + threadIdx.x; // index of thread: block_idx * 896 + thread_idx     // For fullHD: blockIdx € [0,2315] and threadIdx € [0,896] so n € [0, 2315*896[
    int i = n / height;
    int j = n % height;

    if (!(n < nbPixel)) return;

    double pas_x = (x_max - x_min) / width;
    double pas_y = (y_max - y_min) / height;
    double z0_re = i * pas_x + x_min;
    double z0_im = j * pas_y + y_min;
    z0_re *= -1; // Mirror on 0x

    // isSuiteDivergente

    double zi_re = z0_re; // init zi = z0
    double zi_im = z0_im;
    double zi_mod = z0_re * z0_re + z0_im * z0_im;

    double zi_re_temp;
    const unsigned int MAX_ITERATION = 200;
    const double MODULE_MAX = 4.0;

    int nb_iteration = 0;
    while (zi_mod < MODULE_MAX && nb_iteration < MAX_ITERATION) {
        zi_re_temp = zi_re;                             // next zi
        zi_re = zi_re * zi_re - zi_im * zi_im + c_re;   // next zi
        zi_im = 2 * zi_re_temp * zi_im + c_im;          // next zi
        zi_mod = zi_re * zi_re + zi_im * zi_im;   // update zi (squared) module
        nb_iteration++;
    }

    if (zi_mod >= MODULE_MAX) { // On a un module très grand
        d_res[i * height + j] = true;
    }
    else {
        d_res[i * height + j] = false;
    }

    // fin isSuiteDivergente

    if (z0_re == 0) d_res[i *height + j] = -1; // Ox
    if (z0_im == 0) d_res[i * height + j] = -1; // Ox

}

// CPU Version (never executed)
#pragma region CPU Version

void calculate_Julia_CPU(double c_re, double c_im, int** res, int width, int height, double scale, double offset_re, double offset_im) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            res[i][j] = isSuiteDivergente(c_re, c_im, (i - offset_re) * scale, (j - offset_im) * scale) ? 1 : 0; // z0 = new Complexe((i - offset.re) * scale, (j - offset.im) * scale)
        }
    }

    return;
}

bool isSuiteDivergente(double c_re, double c_im, double z0_re, double z0_im) {
    double zi_re = z0_re; // init zi = z0
    double zi_im = z0_im;
    double zi_mod = module(z0_re, z0_im);
    double zi_re_temp;
    const unsigned int MAX_ITERATION = 200;
    const double MODULE_MAX = 4.0;

    int i = 0;
    while (zi_mod < MODULE_MAX && i < MAX_ITERATION) {
        zi_re_temp = zi_re;                             // next zi
        zi_re = zi_re * zi_re - zi_im * zi_im + c_re;   // next zi
        zi_im = 2 * zi_re_temp * zi_im + c_im;          // next zi
        zi_mod = module(zi_re, zi_im);                  // update zi module
        i++;
    }

    if (zi_mod >= MODULE_MAX) { // On a un module très grand
        return true;
    }

    return false;
}

double module(double z_re, double z_im) {
    return sqrt(z_re * z_re + z_im * z_im);
}

#pragma endregion